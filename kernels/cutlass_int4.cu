#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/epilogue/threadblock/fusion/visitors.hpp"
#include "cutlass/gemm/kernel/default_gemm_universal_with_visitor.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>


#define CUTLASS_CHECK(status) \
  TORCH_CHECK(status == cutlass::Status::kSuccess, "cutlass error: ", cutlassGetStatusString(status))


// define common params
using ElementA           = cutlass::int4b_t;
using ElementB           = cutlass::int4b_t;
using ElementAccumulator = int32_t;
using OpClass            = cutlass::arch::OpClassTensorOp;
using ArchTag            = cutlass::arch::Sm80;

// how many elements to load at a time -> load 128-bit = 32 x 4-bit
constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;


// we will do input checks in python. A and B are stored as int8
torch::Tensor int4_mm(torch::Tensor A, torch::Tensor B) {
  int M = A.size(0);
  int K = A.size(1) * 2;
  int N = B.size(1);
  torch::Tensor C = torch::empty({M, N}, A.options().dtype(torch::kInt32));

  // some configs for int4 mma
  // https://github.com/NVIDIA/cutlass/blob/v3.5.1/test/unit/gemm/device/gemm_s4t_s4n_s32t_tensor_op_s32_sm80.cu
  // using default config
  // using ThreadblockShape = cutlass::gemm::GemmShape<128, 256, 128>;
  // using WarpShape        = GemmShape<64, 64, 128>;
  // using InstructionShape = GemmShape<16, 8, 64>;
  // static int const kStages = 3;
  using ElementC = int32_t;
  using Gemm = cutlass::gemm::device::Gemm<
    ElementA, cutlass::layout::RowMajor,    // A matrix
    ElementB, cutlass::layout::ColumnMajor, // B matrix
    ElementC, cutlass::layout::RowMajor,    // C matrix
    ElementAccumulator, OpClass, ArchTag
  >;
  Gemm::Arguments args {
    {M, N, K},
    {reinterpret_cast<ElementA *>(A.data_ptr<int8_t>()), K},
    {reinterpret_cast<ElementB *>(B.data_ptr<int8_t>()), K},
    {C.data_ptr<ElementC>(), N},
    {C.data_ptr<ElementC>(), N},
    {1, 0}  // epilogue
  };
  Gemm gemm_op;
  CUTLASS_CHECK(gemm_op(args));

  return C;
}

// we will do input checks in python. A and B are stored as int8
// this function is based on the following cutlass example
// https://github.com/NVIDIA/cutlass/blob/main/examples/47_ampere_gemm_universal_streamk/ampere_gemm_universal_streamk_broadcast.cu
// also with the help of emitted code from cutlass Python  
torch::Tensor scaled_int4_mm(torch::Tensor A, torch::Tensor B, torch::Tensor row_scale, torch::Tensor col_scale) {
  int M = A.size(0);
  int K = A.size(1) * 2;
  int N = B.size(1);
  torch::Tensor C = torch::empty({M, N}, row_scale.options());

  using ElementC        = cutlass::bfloat16_t;
  using ElementEpilogue = float;

  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;  // 8

  // some configs for int4 mma
  // https://github.com/NVIDIA/cutlass/blob/v3.5.1/test/unit/gemm/device/gemm_s4t_s4n_s32t_tensor_op_s32_sm80.cu
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 256, 128>;
  using WarpShape        = cutlass::gemm::GemmShape<64, 64, 128>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 64>;

  constexpr int numStages = 3;
  constexpr int numEpilogueStages = 1;

  // build epilogue visitor tree
  using OutputTileThreadMap = cutlass::epilogue::threadblock::OutputTileThreadLayout<
    ThreadblockShape, WarpShape, ElementC, AlignmentC, numEpilogueStages
  >;

  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;
  constexpr auto RoundMode = cutlass::FloatRoundStyle::round_to_nearest;
  using Multiply = cutlass::epilogue::threadblock::VisitorCompute<
    cutlass::multiplies, ElementEpilogue, ElementEpilogue, RoundMode
  >;

  // (1, N)
  using ColScale = cutlass::epilogue::threadblock::VisitorRowBroadcast<
    OutputTileThreadMap, ElementC,
    cute::Stride<cute::_0, cute::_1, int32_t>  // MNL
  >;
  using EVTCompute0 = cutlass::epilogue::threadblock::Sm80EVT<Multiply, Accum, ColScale>;

  // (M, 1)
  using RowScale = cutlass::epilogue::threadblock::VisitorColBroadcast<
    OutputTileThreadMap, ElementC,
    cute::Stride<cute::_1, cute::_0, int32_t>  // MNL
  >;
  using EVTCompute1 = cutlass::epilogue::threadblock::Sm80EVT<Multiply, EVTCompute0, RowScale>;

  using Output = cutlass::epilogue::threadblock::VisitorAuxStore<
    OutputTileThreadMap, ElementC, RoundMode,
    cute::Stride<int64_t, cute::_1, int64_t>  // MNL
  >;
  using EVTOutput = cutlass::epilogue::threadblock::Sm80EVT<Output, EVTCompute1>;

  // to make this work with GemmIdentityThreadblockSwizzle, requires the patch from
  // https://github.com/NVIDIA/cutlass/pull/1753
  using EVTKernel = typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
    ElementA, cutlass::layout::RowMajor,    cutlass::ComplexTransform::kNone, AlignmentA,
    ElementB, cutlass::layout::ColumnMajor, cutlass::ComplexTransform::kNone, AlignmentB,
    ElementC, cutlass::layout::RowMajor,                                      AlignmentC,
    ElementAccumulator, ElementEpilogue, OpClass, ArchTag,
    ThreadblockShape, WarpShape, InstructionShape,
    EVTOutput,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<1>,
    numStages,
    cutlass::arch::OpMultiplyAddSaturate,  // OpMultiplyAdd does not work
    numEpilogueStages
  >::GemmKernel;
  using DeviceGemm = cutlass::gemm::device::GemmUniversalAdapter<EVTKernel>;

  const ElementA *A_ptr         = reinterpret_cast<ElementA *>(A.data_ptr<int8_t>());
  const ElementB *B_ptr         = reinterpret_cast<ElementB *>(B.data_ptr<int8_t>());
  const ElementC *col_scale_ptr = reinterpret_cast<ElementC *>(col_scale.data_ptr<torch::BFloat16>());
  const ElementC *row_scale_ptr = reinterpret_cast<ElementC *>(row_scale.data_ptr<torch::BFloat16>());
  ElementC *C_ptr               = reinterpret_cast<ElementC *>(C.data_ptr<torch::BFloat16>());

  typename EVTOutput::Arguments callback_args{
    {
      {
        {},                                                                  // Accum
        {col_scale_ptr, ElementC(0), {cute::_0{}, cute::_1{}, int32_t(N)}},  // ColScale
        {}                                                                   // Multiply
      },                                                                     // EVTCompute0
      {row_scale_ptr, ElementC(0), {cute::_1{}, cute::_0{}, int32_t(M)}},    // RowScale
      {}                                                                     // Multiply
    },                                                                       // EVTCompute1
    {C_ptr, {int64_t{N}, cute::_1{}, int64_t{M*N}}}                          // EVTOutput
  };

  typename DeviceGemm::Arguments args(
    cutlass::gemm::GemmUniversalMode::kGemm,
    cutlass::gemm::GemmCoord{M, N, K},
    1,                              // batch_split
    callback_args,
    A_ptr, B_ptr, nullptr, nullptr, // unsued C_ptr and D_ptr
    M * K, N * K, 0, 0,             // batch_stride A, B, C, D
    K, K, 0, 0                      // stride A, B, C, D
  );

  DeviceGemm gemm_op;
  auto stream = at::cuda::getCurrentCUDAStream();
  CUTLASS_CHECK(gemm_op.can_implement(args));
  CUTLASS_CHECK(gemm_op(args, nullptr, stream));

  return C;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("int4_mm", &int4_mm);
  m.def("scaled_int4_mm", &scaled_int4_mm);
}
