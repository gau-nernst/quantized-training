// CUTLASS example 79

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cutlass/detail/sm100_blockscaled_layout.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/util/packed_stride.hpp"

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#define CUTLASS_CHECK(status) \
  TORCH_CHECK(status == cutlass::Status::kSuccess, "cutlass error: ", cutlassGetStatusString(status))

using namespace cute;

using ElementOutput = cutlass::bfloat16_t;
using ElementScale  = float;
using ElementAcc    = float;

using ArchTag       = cutlass::arch::Sm120;
using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

using TileShape    = Shape<_128, _128, _128>;
using ClusterShape = Shape<_1, _1, _1>;

constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;
// constexpr auto RoundStyle     = cutlass::FloatRoundStyle::round_to_nearest;

torch::Tensor mxfp4_mm(torch::Tensor A, torch::Tensor B, torch::Tensor scales_A, torch::Tensor scales_B)
{
  int M = A.size(0);
  int K = A.size(1) * 2;
  int N = B.size(1);
  torch::Tensor D = torch::empty({M, N}, A.options().dtype(torch::kBFloat16));

  using ElementInput = cutlass::mx_float4_t<cutlass::float_e2m1_t>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAcc, ElementAcc,
      ElementOutput, cutlass::layout::RowMajor, AlignmentOutput,
      ElementOutput, cutlass::layout::RowMajor, AlignmentOutput,
      cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      ElementInput, cutlass::layout::RowMajor, 32,
      ElementInput, cutlass::layout::ColumnMajor, 32,
      ElementAcc,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using Sm1xxBlkScaledConfig = typename GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;
  using DataType = typename ElementInput::DataType;
  using ScaleFactorType = typename ElementInput::ScaleFactorType;

  auto stride_A = cutlass::make_cute_packed_stride(typename GemmKernel::StrideA{}, {M, K, 1});
  auto stride_B = cutlass::make_cute_packed_stride(typename GemmKernel::StrideB{}, {N, K, 1});
  auto stride_D = cutlass::make_cute_packed_stride(typename GemmKernel::StrideD{}, {M, N, 1});

  auto layout_SFA = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
  auto layout_SFB = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, 1},
      {
          reinterpret_cast<DataType *>(A.data_ptr()),
          stride_A,
          reinterpret_cast<DataType *>(B.data_ptr()),
          stride_B,
          reinterpret_cast<ScaleFactorType *>(scales_A.data_ptr()),
          layout_SFA,
          reinterpret_cast<ScaleFactorType *>(scales_B.data_ptr()),
          layout_SFB,
      },
      {
          {1.0f, 0.0f},
          nullptr,
          stride_D,
          reinterpret_cast<ElementOutput *>(D.data_ptr()),
          stride_D,
      }};

  Gemm gemm;
  CUTLASS_CHECK(gemm.can_implement(arguments));

  long workspace_size = Gemm::get_workspace_size(arguments);
  torch::Tensor workspace = torch::empty({workspace_size}, A.options().dtype(torch::kByte));
  auto stream = at::cuda::getCurrentCUDAStream();

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));
  CUTLASS_CHECK(gemm.run(stream));

  return D;
}

torch::Tensor nvfp4_mm(torch::Tensor A, torch::Tensor B, torch::Tensor scales_A, torch::Tensor scales_B, torch::Tensor global_scale)
{
  int M = A.size(0);
  int K = A.size(1) * 2;
  int N = B.size(1);
  torch::Tensor D = torch::empty({M, N}, A.options().dtype(torch::kBFloat16));

  using ElementInput = cutlass::nv_float4_t<cutlass::float_e2m1_t>;

  using EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAcc, ElementAcc,
      ElementOutput, cutlass::layout::RowMajor, AlignmentOutput,
      ElementOutput, cutlass::layout::RowMajor, AlignmentOutput,
      EpilogueScheduleType>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      ElementInput, cutlass::layout::RowMajor, 32,
      ElementInput, cutlass::layout::ColumnMajor, 32,
      ElementAcc,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using Sm1xxBlkScaledConfig = typename GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;
  using DataType = typename ElementInput::DataType;
  using ScaleFactorType = typename ElementInput::ScaleFactorType;

  auto stride_A = cutlass::make_cute_packed_stride(typename GemmKernel::StrideA{}, {M, K, 1});
  auto stride_B = cutlass::make_cute_packed_stride(typename GemmKernel::StrideB{}, {N, K, 1});
  auto stride_D = cutlass::make_cute_packed_stride(typename GemmKernel::StrideD{}, {M, N, 1});

  auto layout_SFA = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
  auto layout_SFB = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));

  auto *A_ptr = reinterpret_cast<const DataType *>(A.data_ptr());
  auto *B_ptr = reinterpret_cast<const DataType *>(B.data_ptr());
  auto *scales_A_ptr = reinterpret_cast<const ScaleFactorType *>(scales_A.data_ptr());
  auto *scales_B_ptr = reinterpret_cast<const ScaleFactorType *>(scales_B.data_ptr());
  float global_scale_f32 = global_scale.item<float>();
  auto *D_ptr = reinterpret_cast<ElementOutput *>(D.data_ptr());

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, 1},
      {
          A_ptr, stride_A,
          B_ptr, stride_B,
          scales_A_ptr, layout_SFA,
          scales_B_ptr, layout_SFB,
      },
      {
        {global_scale_f32, 0.0f},
        D_ptr, stride_D,
        D_ptr, stride_D,
      }};

  Gemm gemm;
  CUTLASS_CHECK(gemm.can_implement(arguments));

  long workspace_size = Gemm::get_workspace_size(arguments);
  torch::Tensor workspace = torch::empty({workspace_size}, A.options().dtype(torch::kByte));
  auto stream = at::cuda::getCurrentCUDAStream();

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));
  CUTLASS_CHECK(gemm.run(stream));

  return D;
}

TORCH_LIBRARY_IMPL(qtrain, CUDA, m)
{
  m.impl("qtrain::mxfp4_mm", &mxfp4_mm);
  m.impl("qtrain::nvfp4_mm", &nvfp4_mm);
}
